#include "hip/hip_runtime.h"
#include "retinaface5/decode.h"
#include "stdio.h"

namespace nvinfer1
{
    DecodePlugin::DecodePlugin(const int cudaThread):thread_count_(cudaThread)
    {
    }
    
    DecodePlugin::~DecodePlugin()
    {
    }
    
    // create the plugin at runtime from a byte stream
    DecodePlugin::DecodePlugin(const void* data, size_t length)
    {
    }

    void DecodePlugin::serialize(void* buffer)
    {
    }
    
    size_t DecodePlugin::getSerializationSize()
    {  
        return 0;
    }

    int DecodePlugin::initialize()
    { 
        return 0;
    }
    
    Dims DecodePlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        //output the result to channel
        int totalCount = 0;
        totalCount += decodeplugin::INPUT_H / 8 * decodeplugin::INPUT_W / 8 * 2 * sizeof(decodeplugin::Detection) / sizeof(float);
        totalCount += decodeplugin::INPUT_H / 16 * decodeplugin::INPUT_W / 16 * 2 * sizeof(decodeplugin::Detection) / sizeof(float);
        totalCount += decodeplugin::INPUT_H / 32 * decodeplugin::INPUT_W / 32 * 2 * sizeof(decodeplugin::Detection) / sizeof(float);

        return Dims3(totalCount + 1, 1, 1);
    }

    __device__ float Logist(float data){ return 1./(1. + exp(-data)); };

    __global__ void CalDetection(const float *input, float *output, int num_elem, int step, int anchor) {
 
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= num_elem) return;

        int h = decodeplugin::INPUT_H / step;
        int w = decodeplugin::INPUT_W / step;
        int y = idx / w;
        int x = idx % w;
        const float *bbox_reg = &input[0];
        const float *cls_reg = &input[2 * 4 * num_elem];
        const float *lmk_reg = &input[2 * 4 * num_elem + 2 * 2 * num_elem];

        for (int k = 0; k < 2; ++k) {
            float conf1 = cls_reg[idx + k * num_elem * 2];
            float conf2 = cls_reg[idx + k * num_elem * 2 + num_elem];
            conf2 = exp(conf2) / (exp(conf1) + exp(conf2));
            if (conf2 <= 0.02) continue;

            float *res_count = output;
            int count = (int)atomicAdd(res_count, 1);
            char* data = (char *)res_count + sizeof(float) + count * sizeof(decodeplugin::Detection);
            decodeplugin::Detection* det = (decodeplugin::Detection*)(data);

            float prior[4];
            prior[0] = ((float)x + 0.5) / w;
            prior[1] = ((float)y + 0.5) / h;
            prior[2] = (float)anchor * (k + 1) / decodeplugin::INPUT_W;
            prior[3] = (float)anchor * (k + 1) / decodeplugin::INPUT_H;

            //Location
            det->bbox[0] = prior[0] + bbox_reg[idx + k * num_elem * 4] * 0.1 * prior[2];
            det->bbox[1] = prior[1] + bbox_reg[idx + k * num_elem * 4 + num_elem] * 0.1 * prior[3];
            det->bbox[2] = prior[2] * exp(bbox_reg[idx + k * num_elem * 4 + num_elem * 2] * 0.2);
            det->bbox[3] = prior[3] * exp(bbox_reg[idx + k * num_elem * 4 + num_elem * 3] * 0.2);
            det->bbox[0] -= det->bbox[2] / 2;
            det->bbox[1] -= det->bbox[3] / 2;
            det->bbox[2] += det->bbox[0];
            det->bbox[3] += det->bbox[1];
            det->bbox[0] *= decodeplugin::INPUT_W;
            det->bbox[1] *= decodeplugin::INPUT_H;
            det->bbox[2] *= decodeplugin::INPUT_W;
            det->bbox[3] *= decodeplugin::INPUT_H;
            det->class_confidence = conf2;
            for (int i = 0; i < 10; i += 2) {
                det->landmark[i] = prior[0] + lmk_reg[idx + k * num_elem * 10 + num_elem * i] * 0.1 * prior[2];
                det->landmark[i+1] = prior[1] + lmk_reg[idx + k * num_elem * 10 + num_elem * (i + 1)] * 0.1 * prior[3];
                det->landmark[i] *= decodeplugin::INPUT_W;
                det->landmark[i+1] *= decodeplugin::INPUT_H;
            }
        }
    }
   
    void DecodePlugin::forwardGpu(const float *const * inputs, float * output, hipStream_t stream, int batchSize) 
    {
        int num_elem = 0;
        int base_step = 8;
        int base_anchor = 16;
        int thread_count;
        hipMemset(output, 0, sizeof(float));
        for (unsigned int i = 0; i < 3; ++i)
        {
            num_elem = decodeplugin::INPUT_H / base_step * decodeplugin::INPUT_W / base_step;
            thread_count = (num_elem < thread_count_) ? num_elem : thread_count_;
            CalDetection<<< (num_elem + thread_count - 1) / thread_count, thread_count>>>
                (inputs[i], output, num_elem, base_step, base_anchor);
            base_step *= 2;
            base_anchor *= 4;
        }
    }

    int DecodePlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        //assert(batchSize == 1);
        //GPU
        //CUDA_CHECK(hipStreamSynchronize(stream));
        forwardGpu((const float *const *)inputs,(float *)outputs[0],stream,batchSize);

        return 0;
    };

}
